#include "hip/hip_runtime.h"
/*******************************************************************
*   CUDALERP.cu
*   CUDALERP
*
*	Author: Kareem Omar
*	kareem.omar@uah.edu
*	https://github.com/komrad36
*
*	Last updated Jan 7, 2016
*******************************************************************/
//
// The file CUDALERP.h exposes two extremely high performance GPU
// resize operations,
// CUDALERP (bilinear interpolation), and 
// CUDANERP (nearest neighbor interpolation), for 8-bit unsigned
// integer (i.e. grayscale) data.
//
// For 32-bit float data, see the CUDAFLERP project instead.
//
// CUDALERP offers superior accuracy to CUDA's built-in texture
// interpolator at comparable performance. The accuracy if compiled
// with -use-fast-math off is nearly equivalent to my CPU interpolator,
// KLERP, while still being as fast as the built-in interpolation.
// 
// Particularly for large images, CUDALERP dramatically outperforms
// even the highly tuned CPU AVX2 versions.
// 
// All functionality is contained in the header 'CUDALERP.h' and
// the source file 'CUDALERP.cu' and has no external dependencies at all.
// 
// Note that these are intended for computer vision use(hence the speed)
// and are designed for grayscale images.
// 
// The file 'main.cpp' is an example and speed test driver.
//

#include "CUDALERP.h"

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(256, 0)
#endif
CUDANERP_kernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, uint8_t* __restrict const d_out, const int neww) {
	uint32_t x = (blockIdx.x << 9) + (threadIdx.x << 1);
	const uint32_t y = blockIdx.y;
	const float fy = y*gys;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fx = x*gxs;
		float res = 255.0f*tex2D<float>(d_img_tex, fx, fy);
		if (x < neww) d_out[y*neww + x] = res;
	}
}

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(256, 0)
#endif
CUDALERP_kernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, uint8_t* __restrict const d_out, const int neww) {
	uint32_t x = (blockIdx.x << 9) + (threadIdx.x << 1);
	const uint32_t y = blockIdx.y;
	const float fy = (y + 0.5f)*gys - 0.5f;
	const float wt_y = fy - floor(fy);
	const float invwt_y = 1.0f - wt_y;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fx = (x + 0.5f)*gxs - 0.5f;
		// less accurate and not really much (or any) faster
		// -----------------
		// const float res = tex2D<float>(d_img_tex, fx, fy);
		// -----------------
		const float4 f = tex2Dgather<float4>(d_img_tex, fx + 0.5f, fy + 0.5f);
		const float wt_x = fx - floor(fx);
		const float invwt_x = 1.0f - wt_x;
		const float xa = invwt_x*f.w + wt_x*f.z;
		const float xb = invwt_x*f.x + wt_x*f.y;
		const float res = 255.0f*(invwt_y*xa + wt_y*xb) + 0.5f;
		// -----------------
		if (x < neww) d_out[y*neww + x] = res;
	}
}

void CUDANERP(const hipTextureObject_t d_img_tex, const int oldw, const int oldh, uint8_t* __restrict const d_out, const uint32_t neww, const uint32_t newh) {
	const float gxs = static_cast<float>(oldw) / static_cast<float>(neww);
	const float gys = static_cast<float>(oldh) / static_cast<float>(newh);
	CUDANERP_kernel<<<{((neww - 1) >> 9) + 1, newh}, 256>>>(d_img_tex, gxs, gys, d_out, neww);
	hipDeviceSynchronize();
}

void CUDALERP(const hipTextureObject_t d_img_tex, const int oldw, const int oldh, uint8_t* __restrict const d_out, const uint32_t neww, const uint32_t newh) {
	const float gxs = static_cast<float>(oldw) / static_cast<float>(neww);
	const float gys = static_cast<float>(oldh) / static_cast<float>(newh);
	CUDALERP_kernel<<<{((neww - 1) >> 9) + 1, newh}, 256>>>(d_img_tex, gxs, gys, d_out, neww);
	hipDeviceSynchronize();
}
